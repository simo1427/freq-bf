//
// Created by HP on 23/05/2024.
//

#include "utils.cuh"

void errorCheck(hipError_t err)
{
    if (err != hipSuccess) {
        std::cerr << "CUDA error:" << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}