#include "hip/hip_runtime.h"
//
// Created by HP on 31/05/2024.
//

#include "fastBilateral.cuh"
#include "utils.cuh"
#include "spatial/separableConvolution.cuh"

#define BF_POPULATE_WIDTH 128
#define BF_POPULATE_HEIGHT 2

#define BF_COLLECT_WIDTH 32
#define BF_COLLECT_HEIGHT 8


#define MAX_COEFS_NUM 20 // TODO: check for a better value, possibly derived from the amount of VRAM

__constant__ float d_Coefs[MAX_COEFS_NUM];
__constant__ float2 d_trigLut[256][MAX_COEFS_NUM];
// TODO: is this an efficient memory layout?
// shouldn't be a problem, as constant memory should take care of that according to the CUDA C++ Programming Guide

void setCoefficients(float* h_Coefs, int n)
{
    hipMemcpyToSymbol(HIP_SYMBOL(d_Coefs), h_Coefs, n);
}

void populateLut(int numberOfCoefficients, float T)
{
    float2 h_trigLut[256][MAX_COEFS_NUM];
    for (int k = 0; k < numberOfCoefficients; k++)
    {
        for (int j = 0; j < 256; j++)
        {
            h_trigLut[j][k].x = cosf(static_cast<float>(j / 255.0f) * 2.0f * M_PI * k / T);
            h_trigLut[j][k].y = sinf(static_cast<float>(j / 255.0f) * 2.0f * M_PI * k / T);
        }
    }

    hipMemcpyToSymbol(HIP_SYMBOL(d_trigLut), h_trigLut, 256 * MAX_COEFS_NUM * sizeof(float2));
}

__global__ void fastBFPopulate(uint8_t* d_Inp, float4* d_Buf, int width, int height, int k, size_t srcPitch, size_t bufPitch)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (y >= height || x >= width)
        return;

    // TODO: shared memory? maybe first pitched memory, then shared...
    uint8_t* d_InpRow = d_Inp + y * srcPitch;

    uint8_t px = d_InpRow[x];
    // TODO: the trick for accessing many uint8_t's at the same time? I saw that recently in an NVIDIA presentation

    float pxScaled = static_cast<float>(px) / 255.0f;

    // order: x:cos y:sin z:cosIntensity w:sinIntensity

    float2 vals = d_trigLut[px][k];
    float4 tmp = make_float4(vals.x, vals.y, pxScaled * vals.x, pxScaled * vals.y);

    float4* d_BufRow = (float4*) ((char*) d_Buf + y * bufPitch);
    d_BufRow[x] = tmp;

}

__global__ void collectResults(float4* d_OutNonSummed, uint8_t* d_Inp,
                               float4* d_Out, int width, int height,
                               int k, size_t inpPitch, size_t bufPitch, size_t outPitch)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (y >= height || x >= width)
        return;

    uint8_t* d_InpRow = d_Inp + y * inpPitch;
    uint8_t px = d_InpRow[x];

    float4* d_OutNonSummedRow = (float4*)((char*) d_OutNonSummed + y * bufPitch);
    float4 tmp = d_OutNonSummedRow[x];

    float4* d_OutRow = (float4*) ((char*)d_Out + y * outPitch);
    float4 oldOut = d_OutRow[x];

    float4 out;
    float2 sinCosVals = d_trigLut[px][k];

    float outX = __fmul_rn(sinCosVals.x, tmp.x);
    float outY = __fmul_rn(sinCosVals.y, tmp.y);
    float outZ = __fmul_rn(sinCosVals.x, tmp.z);
    float outW = __fmul_rn(sinCosVals.y, tmp.w);

    out.x = __fmaf_rn(d_Coefs[k], outX, oldOut.x);
    out.y = __fmaf_rn(d_Coefs[k], outY, oldOut.y);
    out.z = __fmaf_rn(d_Coefs[k], outZ, oldOut.z);
    out.w = __fmaf_rn(d_Coefs[k], outW, oldOut.z);

    d_OutRow[x] = out;
}

__global__ void obtainFinalImage(float4* d_OutSummed,
                               float* d_Out, int width, int height,
                               size_t inpPitch, size_t outPitch)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (y >= height || x >= width)
        return;


    float4* d_OutSummedRow = (float4*) ((char*)d_OutSummed + y * inpPitch);
    float4 tmp = d_OutSummedRow[x];

    float* d_OutRow = (float*) ((char*)d_Out + y * outPitch);
    float sum = __fadd_rn(tmp.w, tmp.z);
    float W = __fadd_rn(tmp.x, tmp.y);
    d_OutRow[x] = __fdiv_rn(sum, W);
}

void debugOutBuf(float4* h_BfBuf, int rows, int cols)
{
    cv::Mat dbgOut = cv::Mat(rows, cols, CV_32F);
    std::string filenames[] = {"./cosImg.tif", "./sinImg.tif", "./cosIntensityImg.tif", "./sinIntensityImg.tif"};

    for (int k = 0; k < 4; k++)
    {
        for (int i = 0; i < rows; i++)
        {
            float* ptrDbgOut = dbgOut.ptr<float>(i);

            union {
                float4 oneWord;
                float fourFloats[4];
            } tmp;

            for (int j = 0; j < cols; j++)
            {
                tmp.oneWord = h_BfBuf[i * cols + j];
                ptrDbgOut[j] = tmp.fourFloats[k];
            }
        }

        cv::imwrite(filenames[k], dbgOut);

    }

    dbgOut.release();
}

void BF_approx_gpu(cv::Mat &input, cv::Mat &output, cv::Mat &spatialKernel, double sigmaRange, range_krn_t rangeKrn, int numberOfCoefficients, float T)
{
    assert(input.type() == CV_8U);

    int width = input.cols;
    int height = input.rows;

    if (numberOfCoefficients == 0)
        // modified heuristic compared to Honours project
        numberOfCoefficients =(int)ceil(1.5 * 2 / (6 * sigmaRange)) + 1;

    auto doubleCoefs = getFourierCoefficients(sigmaRange, T, numberOfCoefficients, rangeKrn);
    std::vector<float> coefsVec{doubleCoefs.begin(), doubleCoefs.end()};



#ifdef DEBUG_PRINT_FOURIER
    std::cout << "Fourier coefs:\n";
    for (int i = 0; i < coefs.size(); i++)
    {
        std::cout << coefs[i] << " ";
    }
    std::cout << std::endl;
#endif

    // Copy the coefficients to constant memory
    setCoefficients(coefsVec.data(), coefsVec.size() * sizeof(float));

    populateLut(numberOfCoefficients, T);
    setConvolutionKernel(spatialKernel.ptr<float>(), spatialKernel.rows);

    // Allocate arrays for intermediate images

    int frameSize = input.rows * input.cols;

    size_t uint8Pitch, floatPitch, float2Pitch, float4Pitch;

    uint8_t* d_Inp;
    checkCudaErrors(hipMallocPitch(&d_Inp, &uint8Pitch,
                                    input.cols * sizeof(uint8_t), input.rows));

    float4* d_OutSummed;
    checkCudaErrors(hipMallocPitch(&d_OutSummed, &float4Pitch,
                                    input.cols * sizeof(float4), input.rows));

    float* d_Out;
    checkCudaErrors(hipMallocPitch(&d_Out, &floatPitch,
                                    input.cols * sizeof(float), input.rows));

    float4* d_BfBuf;
    checkCudaErrors(hipMallocPitch(&d_BfBuf, &float4Pitch,
                                    input.cols * sizeof(float4), input.rows));
    float4* d_OutNonSummed;
    checkCudaErrors(hipMallocPitch(&d_OutNonSummed, &float4Pitch,
                                    input.cols * sizeof(float4), input.rows));
    float4* d_OutNonSummedBuf;
    checkCudaErrors(hipMallocPitch(&d_OutNonSummedBuf, &float4Pitch,
                                    input.cols * sizeof(float4), input.rows));

    // copy the image to the GPU

    checkCudaErrors(hipMemcpy2D(d_Inp, uint8Pitch,
                                 input.ptr<uint8_t>(), input.cols * sizeof(uint8_t),
                                 input.cols * sizeof(uint8_t), input.rows,
                                 hipMemcpyHostToDevice));

    // create events for measuring execution time
    hipEvent_t start, finish;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&finish);

    // execute kernels

    dim3 populateThreads(BF_POPULATE_WIDTH, BF_POPULATE_HEIGHT);
    dim3 populateBlocks(width / populateThreads.y + (width % populateThreads.y ? 1 : 0), height / populateThreads.x + (height % populateThreads.x ? 1 : 0));

    dim3 finalThreads(BF_COLLECT_WIDTH, BF_COLLECT_HEIGHT);
    dim3 finalBlocks(width / populateThreads.y + (width % populateThreads.y ? 1 : 0), height / populateThreads.x + (height % populateThreads.x ? 1 : 0));


    // for debug image output
    float4* h_BfBuf = (float4*) malloc(frameSize * sizeof(float4));

//    printf("Number of coefficients: %d\n", numberOfCoefficients);
    hipEventRecord(start, 0);


    // TODO: enqueue convolutions for each of the images in memory
    for (int i = 0; i < numberOfCoefficients; i++) {
        fastBFPopulate<<<populateBlocks, populateThreads>>>(d_Inp,d_BfBuf,
                                                            width, height,
                                                            i, uint8Pitch, float4Pitch);

        sepFilterf4(d_OutNonSummed,
                    d_BfBuf,
                    d_OutNonSummedBuf,
                    width,
                    height,
                    spatialKernel.rows,
                    float4Pitch);

//        checkCudaErrors(hipMemcpy2D(h_BfBuf, input.cols * sizeof(float4),
//                                     d_OutNonSummed, float4Pitch,
//                                     input.cols * sizeof(float4), input.rows,
//                                     hipMemcpyDeviceToHost));
//        debugOutBuf(h_BfBuf, input.rows, input.cols);

        collectResults<<<finalBlocks, finalThreads>>>(d_OutNonSummed,
                                                            d_Inp, d_OutSummed,
                                                            width, height,
                                                            i, uint8Pitch,
                                                            float4Pitch, float4Pitch);

    }

    obtainFinalImage<<<finalBlocks, finalThreads>>>(d_OutSummed, d_Out, width, height, float4Pitch, floatPitch);

    hipEventRecord(finish, 0); // Beware of streams if they are going to be added later!
    hipEventSynchronize(finish);
    hipEventElapsedTime(&elapsedTime, start, finish);
    // copy result back from the GPU


//    checkCudaErrors(hipMemcpy(output.ptr<float>(), d_OutSummed, frameSize * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy2D(output.ptr<float>(), input.cols * sizeof(float),
                                 d_Out, floatPitch,
                                 input.cols * sizeof(float), input.rows,
                                 hipMemcpyDeviceToHost));


    free(h_BfBuf);

    printf("Elapsed time: %f ms\n", elapsedTime);

    // cleanup
    hipEventDestroy(start);
    hipEventDestroy(finish);
    hipFree(d_Inp);
    hipFree(d_OutSummed);
    hipFree(d_OutNonSummed);
    hipFree(d_BfBuf);
    hipFree(d_OutNonSummedBuf);
    hipFree(d_Out);
    return;

}
